
#include <hip/hip_runtime.h>
#include <__clang_cuda_builtin_vars.h>
#include <stdio.h>

#define TILE_DIM 32
#define BLOCK_SIZE 8
#define mx 2048
#define my 2048

__global__ void transpose(float* odata, float* idata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;
    if( x >= mx || y >= my) return;
    for (int i = 0 ; i < TILE_DIM; i += BLOCK_SIZE)
    {
        odata[x * w + y + i] = idata[(y+i)*w +x];
    }

}


__global__ void transpose2(float* odata, float* idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM];
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;
    if( x >= mx || y >= my) return;
    for (int i = 0 ; i < TILE_DIM; i += BLOCK_SIZE)
    {
        tile[threadIdx.y + i][threadIdx.x] = idata[(y + i) * w + x];
    }
    __syncthreads();
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for( int i = 0; i < TILE_DIM; i += BLOCK_SIZE){
        odata[(y + i) * w + x] = tile[threadIdx.x][threadIdx.y + i];
    }
}

bool check(float *c_cpu, float* c_gpu)
{
    for (int r = 0; r < mx; r++){
        for (int c = 0; c < my; c++){
            if(c_cpu[r * mx + c] != c_gpu[r * my + c]){
                return false;
            }
        }
    }
    return true;
}


int main() 
{
    size_t size = mx * my * sizeof(float);
    float *h_idata, *h_odata, *d_idata, *d_odata, *res;
    hipHostMalloc(&h_idata, size, hipHostMallocDefault);
    hipHostMalloc(&h_odata, size, hipHostMallocDefault);
    hipHostMalloc(&res, size, hipHostMallocDefault);
    hipMalloc(&d_idata, size);
    hipMalloc(&d_odata, size);
    dim3 threads(TILE_DIM, BLOCK_SIZE, 1);
    dim3 blocks((mx+TILE_DIM-1) / TILE_DIM , (my+TILE_DIM-1) / TILE_DIM ,1 );
    for(int i = 0; i < mx; i++)
    {
        for(int j = 0; j < my; j++)
        {
            h_idata[i*my+j] = i * my + j;
            res[i * my + j] = j * my + i;
        }
    }

    hipEvent_t startEvent, stopEvent;
    float ms;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);
    hipEventRecord(startEvent,0);
    for(int i = 0 ; i < 100; i++)
        transpose<<<blocks, threads>>>(d_odata, d_idata);
    hipEventRecord(stopEvent,0);
    transpose<<<blocks, threads>>>(d_odata, d_idata);
    hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    check(res, h_odata) ? printf("ok") : printf("error");
    hipHostFree(h_idata);
    hipHostFree(h_odata);
    hipHostFree(res);
    hipFree(d_idata);
    hipFree(d_odata);
}