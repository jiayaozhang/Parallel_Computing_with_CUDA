
#include <hip/hip_runtime.h>
#include <cstdio>

int main()
{
    /*
    get device info:
    */
    int id;
    hipGetDevice(&id);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, id);

    printf(
        "device, id: %d, sms: %d, capability major: %d, capability minor: %d, warp size: %d\n", 
        id, props.multiProcessorCount, props.major, props.minor, props.warpSize
    );
    return 0;
}