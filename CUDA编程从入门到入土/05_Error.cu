
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cassert>

int main()
{
	const int N = -1;
    int *a;
    hipMallocManaged(&a, N * sizeof(int));

    /*
    cuda error:
    */
    hipError_t err;
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("%s\n", hipGetErrorString(err));
        assert(err == hipSuccess);
    }

    hipFree(a);
	return 0;
}