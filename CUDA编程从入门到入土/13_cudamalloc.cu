
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_DIM 32
#define BLOCK_SIZE 8
#define mx 2048
#define my 2048

__global__ void transpose(float* odata, float* idata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;
    if( x >= mx || y >= my) return;
    for (int i = 0 ; i < TILE_DIM; i += BLOCK_SIZE)
    {
        odata[x * w + y + i] = idata[(y + i) * w + x];
    }
}

bool check(float *c_cpu, float* c_gpu)
{
    for (int r = 0; r < mx; r++){
        for (int c = 0; c < my; c++){
            if(c_cpu[r * mx + c] != c_gpu[r * my + c]){
                return false;
            }
        }
    }
    return true;
}


int main() 
{
    size_t size = mx * my * sizeof(float);
    float *h_idata, *h_odata, *d_idata, *d_odata, *res;
    hipHostMalloc(&h_idata, size, hipHostMallocDefault);
    hipHostMalloc(&h_odata, size, hipHostMallocDefault);
    hipHostMalloc(&res, size, hipHostMallocDefault);
    hipMalloc(&d_idata, size);
    hipMalloc(&d_odata, size);
    dim3 threads(TILE_DIM, BLOCK_SIZE, 1);
    dim3 blocks((mx+TILE_DIM-1) / TILE_DIM , (my+TILE_DIM-1) / TILE_DIM ,1 );
    for(int i = 0; i < mx; i++)
    {
        for(int j = 0; j < my; j++)
        {
            h_idata[i*my+j] = i * my + j;
            res[i * my + j] = j * my + i;
        }
    }

    hipMemcpy(d_idata, h_idata, size, hipMemcpyHostToDevice);
    transpose<<<blocks, threads>>>(d_odata, d_idata);
    hipMemcpy(h_odata, d_odata, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    check(res, h_odata) ? printf("ok") : printf("error");
    hipHostFree(h_idata);
    hipHostFree(h_odata);
    hipHostFree(res);
    hipFree(d_idata);
    hipFree(d_odata);
}